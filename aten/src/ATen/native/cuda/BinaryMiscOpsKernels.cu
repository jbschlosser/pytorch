#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void smooth_l1_kernel_cuda(TensorIterator& iter, double beta, bool huber) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "smooth_l1_cuda", [&iter, beta, huber]() {
    scalar_t beta_val(beta);
    gpu_kernel(iter, [beta_val, huber]GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      auto z = ::abs(a - b);
      if (z < beta_val) {
        const auto output = scalar_t(0.5) * z * z;
        return huber ? output : (output / beta_val);
      } else {
        const auto output = z - scalar_t(0.5) * beta_val;
        return huber ? (output * beta_val) : output;
      }
    });
  });
}


void mse_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "mse_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto diff = a - b;
      return diff * diff;
    });
  });
}

REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda);

// DO NOT ADD ANY NEW KERNELS HERE
// CUDA compilation times grow quickly.  It's perfectly acceptable to have a file per kernel.

}} // namespace at::native
